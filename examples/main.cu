#include <iostream>
#include "planners/KGMT.cuh"
int main(void)
{
    // float h_initial[SAMPLE_DIM] = {.5, .5, 0.0, 0.0, 0.0, 0.0, 0.0}, h_goal[SAMPLE_DIM] = {2, 18, 0.0, 0.0, 0.0, 0.0, 0.0};
    float h_initial[SAMPLE_DIM] = {0.3, 0.02, 0.02, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0},
          h_goal[SAMPLE_DIM]    = {.7, .95, .9, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0};
    KGMT kgmt;

    // TODO: clean this up
    int numObstacles = 1;
    float* d_obstacles;
    std::vector<float> obstacles = readObstaclesFromCSV("../include/config/obstacles/pillars/obstacles.csv", numObstacles, DIM);
    hipMalloc(&d_obstacles, numObstacles * 2 * DIM * sizeof(float));
    hipMemcpy(d_obstacles, obstacles.data(), numObstacles * 2 * DIM * sizeof(float), hipMemcpyHostToDevice);

    kgmt.plan(h_initial, h_goal, d_obstacles, numObstacles);

    hipFree(d_obstacles);
    return 0;
    return 0;
}