#include <iostream>
#include "planners/KGMT.cuh"

int main(void)
{
    // --- Remove Previous Bench Data ---
    system("rm -rf Data/*");

    float h_initial[SAMPLE_DIM] = {0.0, 0.02, 0.02, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0},
          h_goal[SAMPLE_DIM]    = {.9, .95, .9, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0};
    KGMT kgmt;

    int numObstacles;
    float* d_obstacles;

    // --- Load Workspace Obstacles ---
    std::vector<float> obstacles = readObstaclesFromCSV("../include/config/obstacles/narrowPassage/obstacles.csv", numObstacles, DIM);

    // --- Transfer Obstacles to device ---
    hipMalloc(&d_obstacles, numObstacles * 2 * DIM * sizeof(float));
    hipMemcpy(d_obstacles, obstacles.data(), numObstacles * 2 * DIM * sizeof(float), hipMemcpyHostToDevice);

    // --- Execute planner ---
    kgmt.planBench(h_initial, h_goal, d_obstacles, numObstacles, 0);

    // --- Free memory ---
    hipFree(d_obstacles);
    return 0;
}