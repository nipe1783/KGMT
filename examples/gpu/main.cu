#include <iostream>
#include "planners/KGMT.cuh"

int main(void)
{
    // --- Remove Previous Bench Data ---
    system("rm -rf Data/*");

    // float h_initial[SAMPLE_DIM] = {.100, .080, .05, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0},
    //       h_goal[SAMPLE_DIM]    = {.800, .950, .900, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0};
    float h_initial[SAMPLE_DIM] = {10.0, 8, 5, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0},
          h_goal[SAMPLE_DIM]    = {80, 95.0, 90.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0};
    KGMT kgmt;

    int numObstacles;
    float* d_obstacles;

    // --- Load Workspace Obstacles ---
    std::vector<float> obstacles = readObstaclesFromCSV("../include/config/obstacles/quadHouse/obstacles.csv", numObstacles, W_DIM);

    // --- Transfer Obstacles to device ---
    hipMalloc(&d_obstacles, numObstacles * 2 * W_DIM * sizeof(float));
    hipMemcpy(d_obstacles, obstacles.data(), numObstacles * 2 * W_DIM * sizeof(float), hipMemcpyHostToDevice);

    // --- Execute planner ---
    kgmt.planBench(h_initial, h_goal, d_obstacles, numObstacles, 0);

    // --- Free memory ---
    hipFree(d_obstacles);
    return 0;
}