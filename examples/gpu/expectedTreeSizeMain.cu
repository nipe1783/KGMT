#include <iostream>
#include "planners/KGMT.cuh"
int main(void)
{
    // --- Remove Previous Bench Data ---
    system("rm -rf Data/*");

    float h_initial[SAMPLE_DIM] = {10.0, 8, 5, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0},
          h_goal[SAMPLE_DIM]    = {80, 95.0, 90.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0};
    // float h_initial[SAMPLE_DIM] = {.100, .80, .05, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0},
    //       h_goal[SAMPLE_DIM]    = {.800, .950, .900, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0};
    KGMT kgmt(100000);

    int numObstacles;
    float* d_obstacles;

    // --- Load Workspace Obstacles ---
    std::vector<float> obstacles = readObstaclesFromCSV("../include/config/obstacles/quadTrees/obstacles.csv", numObstacles, W_DIM);

    // --- Transfer Obstacles to device ---
    hipMalloc(&d_obstacles, numObstacles * 2 * W_DIM * sizeof(float));
    hipMemcpy(d_obstacles, obstacles.data(), numObstacles * 2 * W_DIM * sizeof(float), hipMemcpyHostToDevice);

    // --- Execute planner N times ---
    int N = 50;
    for(int i = 0; i < N; i++)
        {
            // --- Execute planner ---
            kgmt.planBench(h_initial, h_goal, d_obstacles, numObstacles, i);
        }

    // --- Free memory ---
    hipFree(d_obstacles);
    return 0;
}