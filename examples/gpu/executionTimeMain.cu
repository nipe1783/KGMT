#include <iostream>
#include "planners/KGMT.cuh"
int main(void)
{
    // --- Remove Previous Bench Data ---
    system("rm -rf Data/*");

    float h_initial[SAMPLE_DIM] = {30.0, 2.0, 2.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0},
          h_goal[SAMPLE_DIM]    = {70.0, 95.0, 90.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0};
    // float h_initial[SAMPLE_DIM] = {0.3, 0.02, 0.02, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0},
    //       h_goal[SAMPLE_DIM]    = {0.7, .95, .9, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0};
    KGMT kgmt;

    int numObstacles;
    float* d_obstacles;

    // --- Load Workspace Obstacles ---
    std::vector<float> obstacles = readObstaclesFromCSV("../include/config/obstacles/quadTrees/obstacles.csv", numObstacles, W_DIM);

    // --- Transfer Obstacles to device ---
    hipMalloc(&d_obstacles, numObstacles * 2 * W_DIM * sizeof(float));
    hipMemcpy(d_obstacles, obstacles.data(), numObstacles * 2 * W_DIM * sizeof(float), hipMemcpyHostToDevice);

    // --- Execute planner N times ---
    int N = 1000;
    for(int i = 0; i < N; i++)
        {
            // --- Execute planner ---
            kgmt.plan(h_initial, h_goal, d_obstacles, numObstacles);
        }

    // --- Free memory ---
    hipFree(d_obstacles);
    return 0;
}