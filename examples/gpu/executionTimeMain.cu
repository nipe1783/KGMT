#include <iostream>
#include "planners/KGMT.cuh"

int main(void)
{
    // --- Remove Previous Bench Data ---
    system("rm -rf Data/*");

    float h_initial[SAMPLE_DIM] = {10.0, 8, 5, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0},
          h_goal[SAMPLE_DIM]    = {80, 95.0, 90.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0};

    KGMT kgmt;

    int numObstacles;
    float* d_obstacles;

    // --- Load Workspace Obstacles ---
    std::vector<float> obstacles = readObstaclesFromCSV("../include/config/obstacles/quadHouse/obstacles.csv", numObstacles, W_DIM);

    // --- Transfer Obstacles to device ---
    hipMalloc(&d_obstacles, numObstacles * 2 * W_DIM * sizeof(float));
    hipMemcpy(d_obstacles, obstacles.data(), numObstacles * 2 * W_DIM * sizeof(float), hipMemcpyHostToDevice);

    // --- Execute planner N times ---
    int N = 50;
    hipEvent_t start, stop;
    float milliseconds = 0;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    for(int i = 0; i < N; i++)
        {
            hipEventRecord(start);

            // --- Execute planner ---
            kgmt.plan(h_initial, h_goal, d_obstacles, numObstacles);

            hipEventRecord(stop);
            hipEventSynchronize(stop);

            hipEventElapsedTime(&milliseconds, start, stop);
            std::cout << "Execution time: " << milliseconds / 1000.0 << " seconds" << std::endl;
        }

    // --- Free memory ---
    hipFree(d_obstacles);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
