#include "hip/hip_runtime.h"

#include "statePropagator/statePropagator.cuh"

__device__ bool propagateAndCheck(float* x0, float* x1, hiprandState* seed, float* obstacles, int obstaclesCount)
{
    PropagateAndCheckFunc func = getPropagateAndCheckFunc();
    return func ? func(x0, x1, seed, obstacles, obstaclesCount) : false;
}

/***************************/
/* UNICYCLE PROPAGATION FUNCTION */
/***************************/
__device__ bool propagateAndCheckUnicycle(float* x0, float* x1, hiprandState* seed, float* obstacles, int obstaclesCount)
{
    float a        = UNI_MIN_ACC + hiprand_uniform(seed) * (UNI_MAX_ACC - UNI_MIN_ACC);
    float steering = UNI_MIN_STEERING + hiprand_uniform(seed) * (UNI_MAX_STEERING - UNI_MIN_STEERING);
    float duration = UNI_MIN_DT + hiprand_uniform(seed) * (UNI_MAX_DT - UNI_MIN_DT);
    float dt       = duration / NUM_DISC;

    float x     = x0[0];
    float y     = x0[1];
    float theta = x0[2];
    float v     = x0[3];

    float cosTheta, sinTheta, tanSteering;
    float bbMin[DIM], bbMax[DIM];

    bool motionValid = true;
    for(int i = 0; i < NUM_DISC; i++)
        {
            float x0State[DIM] = {x, y};
            cosTheta           = cos(theta);
            sinTheta           = sin(theta);
            tanSteering        = tan(steering);

            // --- State Propagation ---
            x += v * cosTheta * dt;
            y += v * sinTheta * dt;
            theta += v * tanSteering * dt;
            v += a * dt;
            float x1State[DIM] = {x, y};

            // --- Workspace Limit Check ---
            if(x < 0 || x > WS_SIZE || y < 0 || y > WS_SIZE)
                {
                    motionValid = false;
                    break;
                }

            // --- Obstacle Collision Check ---
            for(int d = 0; d < DIM; d++)
                {
                    if(x0State[d] > x1State[d])
                        {
                            bbMin[d] = x0State[d];
                            bbMax[d] = x1State[d];
                        }
                    else
                        {
                            bbMin[d] = x1State[d];
                            bbMax[d] = x0State[d];
                        }
                }
            motionValid = isMotionValid(x0State, x1State, bbMin, bbMax, obstacles, obstaclesCount);
            if(!motionValid) break;
        }

    return true;
}

/***************************/
/* DUBINS PROPAGATION FUNCTION */
/***************************/
__device__ bool propagateAndCheckDubins(float* x0, float* x1, hiprandState* seed, float* obstacles, int obstaclesCount)
{
    printf("/***************************/\n");
    printf("/* DUBINS: TODO */\n");
    printf("/***************************/\n");
    // TODO: Implement Dubins
    return true;
}

/***************************/
/* GET PROPAGATION FUNCTION */
/***************************/
__device__ PropagateAndCheckFunc getPropagateAndCheckFunc()
{
    switch(MODEL)
        {
            case 0:
                return propagateAndCheckUnicycle;
            case 1:
                return propagateAndCheckDubins;
            default:
                return nullptr;
        }
}