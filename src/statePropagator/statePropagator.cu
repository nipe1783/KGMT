#include "hip/hip_runtime.h"

#include "statePropagator/statePropagator.cuh"

__device__ bool propagateAndCheck(float* x0, float* x1, hiprandState* seed, float* obstacles, int obstaclesCount)
{
    PropagateAndCheckFunc func = getPropagateAndCheckFunc();
    return func ? func(x0, x1, seed, obstacles, obstaclesCount) : false;
}

/***************************/
/* UNICYCLE PROPAGATION FUNCTION */
/***************************/
__device__ bool propagateAndCheckUnicycle(float* x0, float* x1, hiprandState* seed, float* obstacles, int obstaclesCount)
{
    float a                 = UNI_MIN_ACC + hiprand_uniform(seed) * (UNI_MAX_ACC - UNI_MIN_ACC);
    float steering          = UNI_MIN_STEERING + hiprand_uniform(seed) * (UNI_MAX_STEERING - UNI_MIN_STEERING);
    float duration          = UNI_MIN_DT + hiprand_uniform(seed) * (UNI_MAX_DT - UNI_MIN_DT);
    int propagationDuration = 1 + (int)(hiprand_uniform(seed) * (MAX_PROPAGATION_DURATION));

    float x     = x0[0];
    float y     = x0[1];
    float theta = x0[2];
    float v     = x0[3];

    float cosTheta, sinTheta, tanSteering;
    float bbMin[DIM], bbMax[DIM];

    bool motionValid = true;
    for(int i = 0; i < propagationDuration; i++)
        {
            float x0State[DIM] = {x, y};
            cosTheta           = cos(theta);
            sinTheta           = sin(theta);
            tanSteering        = tan(steering);

            // --- State Propagation ---
            x += v * cosTheta * STEP_SIZE;
            y += v * sinTheta * STEP_SIZE;
            theta += (v / UNI_LENGTH) * tanSteering * STEP_SIZE;
            v += a * STEP_SIZE;
            float x1State[DIM] = {x, y};

            // --- Workspace Limit Check ---
            if(x < 0 || x > WS_SIZE || y < 0 || y > WS_SIZE)
                {
                    motionValid = false;
                    break;
                }

            // --- Obstacle Collision Check ---
            for(int d = 0; d < DIM; d++)
                {
                    if(x0State[d] > x1State[d])
                        {
                            bbMin[d] = x1State[d];
                            bbMax[d] = x0State[d];
                        }
                    else
                        {
                            bbMin[d] = x0State[d];
                            bbMax[d] = x1State[d];
                        }
                }

            motionValid = motionValid && isMotionValid(x0State, x1State, bbMin, bbMax, obstacles, obstaclesCount);
            if(!motionValid) break;
        }

    x1[0] = x, x1[1] = y, x1[2] = theta, x1[3] = v, x1[4] = a, x1[5] = steering, x1[6] = duration;
    return motionValid;
}

/***************************/
/* DOUBLE INTEGRATOR PROPAGATION FUNCTION */
/***************************/
__device__ bool propagateAndCheckDoubleIntRungeKutta(float* x0, float* x1, hiprandState* seed, float* obstacles, int obstaclesCount)
{
    float ax                = DI_MIN_ACC + hiprand_uniform(seed) * (DI_MAX_ACC - DI_MIN_ACC);
    float ay                = DI_MIN_ACC + hiprand_uniform(seed) * (DI_MAX_ACC - DI_MIN_ACC);
    float az                = DI_MIN_ACC + hiprand_uniform(seed) * (DI_MAX_ACC - DI_MIN_ACC);
    int propagationDuration = 1 + (int)(hiprand_uniform(seed) * (MAX_PROPAGATION_DURATION));

    float x  = x0[0];
    float y  = x0[1];
    float z  = x0[2];
    float vx = x0[3];
    float vy = x0[4];
    float vz = x0[5];

    bool motionValid = true;
    float bbMin[DIM], bbMax[DIM];
    for(int i = 0; i < propagationDuration; i++)
        {
            float x0State[DIM] = {x, y, z};

            // --- State Propagation. 4th order Runge Kutta ---
            x += (vx + (vx + 2 * (vx + ax * STEP_SIZE / 2) + (vx + ax * STEP_SIZE))) * STEP_SIZE / 6;
            y += (vy + (vy + 2 * (vy + ay * STEP_SIZE / 2) + (vy + ay * STEP_SIZE))) * STEP_SIZE / 6;
            z += (vz + (vz + 2 * (vz + az * STEP_SIZE / 2) + (vz + az * STEP_SIZE))) * STEP_SIZE / 6;
            vx += (ax + 2 * ax + 2 * ax + ax) * STEP_SIZE / 6;
            vy += (ay + 2 * ay + 2 * ay + ay) * STEP_SIZE / 6;
            vz += (az + 2 * az + 2 * az + az) * STEP_SIZE / 6;

            // --- Dyanmics Validity Check ---
            if(vx < DI_MIN_VEL || vx > DI_MAX_VEL || vy < DI_MIN_VEL || vy > DI_MAX_VEL || vz < DI_MIN_VEL || vz > DI_MAX_VEL)
                {
                    motionValid = false;
                    break;
                }

            float x1State[DIM] = {x, y, z};

            // --- Workspace Limit Check ---
            if(x < 0 || x > WS_SIZE || y < 0 || y > WS_SIZE || z < 0 || z > WS_SIZE)
                {
                    motionValid = false;
                    break;
                }

            // --- Obstacle Collision Check ---
            for(int d = 0; d < DIM; d++)
                {
                    if(x0State[d] > x1State[d])
                        {
                            bbMin[d] = x1State[d];
                            bbMax[d] = x0State[d];
                        }
                    else
                        {
                            bbMin[d] = x0State[d];
                            bbMax[d] = x1State[d];
                        }
                }

            motionValid = motionValid && isMotionValid(x0State, x1State, bbMin, bbMax, obstacles, obstaclesCount);
            if(!motionValid) break;
        }

    x1[0] = x, x1[1] = y, x1[2] = z, x1[3] = vx, x1[4] = vy, x1[5] = vz, x1[6] = ax, x1[7] = ay, x1[8] = az,
    x1[9] = STEP_SIZE * propagationDuration;
    return motionValid;
}

/***************************/
/* DUBINS AIRPLANE PROPAGATION FUNCTION */
/***************************/
__device__ bool propagateAndCheckDubinsAirplaneRungeKutta(float* x0, float* x1, hiprandState* seed, float* obstacles, int obstaclesCount)
{
    float a                 = DUBINS_AIRPLANE_MIN_ACC + hiprand_uniform(seed) * (DUBINS_AIRPLANE_MAX_ACC - DUBINS_AIRPLANE_MIN_ACC);
    float yawRate           = DUBINS_AIRPLANE_MIN_YR + hiprand_uniform(seed) * (DUBINS_AIRPLANE_MAX_YR - DUBINS_AIRPLANE_MIN_YR);
    float pitchRate         = DUBINS_AIRPLANE_MIN_PR + hiprand_uniform(seed) * (DUBINS_AIRPLANE_MAX_PR - DUBINS_AIRPLANE_MIN_PR);
    int propagationDuration = 1 + (int)(hiprand_uniform(seed) * (MAX_PROPAGATION_DURATION));

    float x     = x0[0];
    float y     = x0[1];
    float z     = x0[2];
    float yaw   = x0[3];
    float pitch = x0[4];
    float v     = x0[5];

    bool motionValid = true;
    float bbMin[DIM], bbMax[DIM];

    for(int i = 0; i < propagationDuration; i++)
        {
            float x0State[DIM] = {x, y, z};

            // --- State Propagation using 4th Order Runge-Kutta Method ---
            x +=
              (STEP_SIZE / 6.0f) *
              (v * cosf(pitch) * cosf(yaw) +
               2.0f * ((v + 0.5f * STEP_SIZE * a) * cosf(pitch + 0.5f * STEP_SIZE * pitchRate) * cosf(yaw + 0.5f * STEP_SIZE * yawRate) +
                       (v + 0.5f * STEP_SIZE * a) * cosf(pitch + 0.5f * STEP_SIZE * pitchRate) * cosf(yaw + 0.5f * STEP_SIZE * yawRate)) +
               (v + STEP_SIZE * a) * cosf(pitch + STEP_SIZE * pitchRate) * cosf(yaw + STEP_SIZE * yawRate));
            y +=
              (STEP_SIZE / 6.0f) *
              (v * cosf(pitch) * sinf(yaw) +
               2.0f * ((v + 0.5f * STEP_SIZE * a) * cosf(pitch + 0.5f * STEP_SIZE * pitchRate) * sinf(yaw + 0.5f * STEP_SIZE * yawRate) +
                       (v + 0.5f * STEP_SIZE * a) * cosf(pitch + 0.5f * STEP_SIZE * pitchRate) * sinf(yaw + 0.5f * STEP_SIZE * yawRate)) +
               (v + STEP_SIZE * a) * cosf(pitch + STEP_SIZE * pitchRate) * sinf(yaw + STEP_SIZE * yawRate));
            z += (STEP_SIZE / 6.0f) * (v * sinf(pitch) +
                                       2.0f * ((v + 0.5f * STEP_SIZE * a) * sinf(pitch + 0.5f * STEP_SIZE * pitchRate) +
                                               (v + 0.5f * STEP_SIZE * a) * sinf(pitch + 0.5f * STEP_SIZE * pitchRate)) +
                                       (v + STEP_SIZE * a) * sinf(pitch + STEP_SIZE * pitchRate));
            yaw += STEP_SIZE * yawRate;
            pitch += STEP_SIZE * pitchRate;
            v += (STEP_SIZE / 6.0f) * (a + 2.0f * (a + a) + a);

            // --- Dynamics Validity Check ---'
            if(v < DUBINS_AIRPLANE_MIN_VEL || v > DUBINS_AIRPLANE_MAX_VEL)
                {
                    motionValid = false;
                    break;
                }
            else if(pitch < DUBINS_AIRPLANE_MIN_PITCH || pitch > DUBINS_AIRPLANE_MAX_PITCH)
                {
                    motionValid = false;
                    break;
                }

            float x1State[DIM] = {x, y, z};

            // --- Workspace Limit Check ---
            if(x < 0 || x > WS_SIZE || y < 0 || y > WS_SIZE || z < 0 || z > WS_SIZE)
                {
                    motionValid = false;
                    break;
                }

            // --- Obstacle Collision Check ---
            for(int d = 0; d < DIM; d++)
                {
                    if(x0State[d] > x1State[d])
                        {
                            bbMin[d] = x1State[d];
                            bbMax[d] = x0State[d];
                        }
                    else
                        {
                            bbMin[d] = x0State[d];
                            bbMax[d] = x1State[d];
                        }
                }

            motionValid = motionValid && isMotionValid(x0State, x1State, bbMin, bbMax, obstacles, obstaclesCount);
            if(!motionValid) break;
        }

    x1[0] = x;
    x1[1] = y;
    x1[2] = z;
    x1[3] = yaw;
    x1[4] = pitch;
    x1[5] = v;
    x1[6] = yawRate;
    x1[7] = pitchRate;
    x1[8] = a;
    x1[9] = STEP_SIZE * propagationDuration;

    return motionValid;
}

/***************************/
/* QUAD COPTER PROPAGATION FUNCTION */
/***************************/
__device__ bool propagateAndCheckQuadRungeKutta(float* x0, float* x1, hiprandState* seed, float* obstacles, int obstaclesCount)
{
    float Zc = QUAD_MIN_Zc + hiprand_uniform(seed) * (QUAD_MAX_Zc - QUAD_MIN_Zc);
    float Lc = QUAD_MIN_Lc + hiprand_uniform(seed) * (QUAD_MAX_Lc - QUAD_MIN_Lc);
    float Mc = QUAD_MIN_Mc + hiprand_uniform(seed) * (QUAD_MAX_Mc - QUAD_MIN_Mc);
    float Nc = QUAD_MIN_Nc + hiprand_uniform(seed) * (QUAD_MAX_Nc - QUAD_MIN_Nc);

    int propagationDuration = 1 + (int)(hiprand_uniform(seed) * (MAX_PROPAGATION_DURATION));

    float x     = x0[0];
    float y     = x0[1];
    float z     = x0[2];
    float phi   = x0[3];
    float theta = x0[4];
    float psi   = x0[5];
    float u     = x0[6];
    float v     = x0[7];
    float w     = x0[8];
    float p     = x0[9];
    float q     = x0[10];
    float r     = x0[11];

    bool motionValid = true;
    float bbMin[W_DIM], bbMax[W_DIM];

    float* h1 = new float[12];
    float* h2 = new float[12];
    float* h3 = new float[12];
    float* h4 = new float[12];

    for(int i = 0; i < propagationDuration; i++)
        {
            float x0State[W_DIM] = {x, y, z};

            ode(h1, x0, Zc, Lc, Mc, Nc);
            ode(h2, h1, Zc, Lc, Mc, Nc);
            ode(h3, h2, Zc, Lc, Mc, Nc);
            ode(h4, h3, Zc, Lc, Mc, Nc);

            x += STEP_SIZE / 6 * (h1[0] + 2.0f * h2[0] + 2.0f * h3[0] + h4[0]);
            y += STEP_SIZE / 6 * (h1[1] + 2.0f * h2[1] + 2.0f * h3[1] + h4[1]);
            z += STEP_SIZE / 6 * (h1[2] + 2.0f * h2[2] + 2.0f * h3[2] + h4[2]);
            phi += STEP_SIZE / 6 * (h1[3] + 2.0f * h2[3] + 2.0f * h3[3] + h4[3]);
            theta += STEP_SIZE / 6 * (h1[4] + 2.0f * h2[4] + 2.0f * h3[4] + h4[4]);
            psi += STEP_SIZE / 6 * (h1[5] + 2.0f * h2[5] + 2.0f * h3[5] + h4[5]);
            u += STEP_SIZE / 6 * (h1[6] + 2.0f * h2[6] + 2.0f * h3[6] + h4[6]);
            v += STEP_SIZE / 6 * (h1[7] + 2.0f * h2[7] + 2.0f * h3[7] + h4[7]);
            w += STEP_SIZE / 6 * (h1[8] + 2.0f * h2[8] + 2.0f * h3[8] + h4[8]);
            p += STEP_SIZE / 6 * (h1[9] + 2.0f * h2[9] + 2.0f * h3[9] + h4[9]);
            q += STEP_SIZE / 6 * (h1[10] + 2.0f * h2[10] + 2.0f * h3[10] + h4[10]);
            r += STEP_SIZE / 6 * (h1[11] + 2.0f * h2[11] + 2.0f * h3[11] + h4[11]);

            float x1State[W_DIM] = {x, y, z};

            // --- Vehicle Dynamics Check ---
            if(u < QUAD_MIN_VEL || u > QUAD_MAX_VEL || v < QUAD_MIN_VEL || v > QUAD_MAX_VEL || w < QUAD_MIN_VEL || w > QUAD_MAX_VEL)
                {
                    motionValid = false;
                    break;
                }

            // --- Workspace Limit Check ---
            if(x < 0 || x > WS_SIZE || y < 0 || y > WS_SIZE || z < 0 || z > WS_SIZE)
                {
                    motionValid = false;
                    break;
                }

            // --- Obstacle Collision Check ---
            for(int d = 0; d < W_DIM; d++)
                {
                    if(x0State[d] > x1State[d])
                        {
                            bbMin[d] = x1State[d];
                            bbMax[d] = x0State[d];
                        }
                    else
                        {
                            bbMin[d] = x0State[d];
                            bbMax[d] = x1State[d];
                        }
                }

            motionValid = motionValid && isMotionValid(x0State, x1State, bbMin, bbMax, obstacles, obstaclesCount);
            if(!motionValid) break;
        }

    delete h1;
    delete h2;
    delete h3;
    delete h4;

    x1[0]  = x;
    x1[1]  = y;
    x1[2]  = z;
    x1[3]  = phi;
    x1[4]  = theta;
    x1[5]  = psi;
    x1[6]  = u;
    x1[7]  = v;
    x1[8]  = w;
    x1[9]  = p;
    x1[10] = q;
    x1[11] = r;
    x1[12] = Zc;
    x1[13] = Lc;
    x1[14] = Mc;
    x1[15] = Nc;
    x1[16] = STEP_SIZE * propagationDuration;

    return motionValid;
}

__device__ void ode(float* x0dot, float* x0, float Zc, float Lc, float Mc, float Nc)
{
    float x, y, z, phi, theta, psi, u, v, w, p, q, r;
    x     = x0[0];
    y     = x0[1];
    z     = x0[2];
    phi   = x0[3];
    theta = x0[4];
    psi   = x0[5];
    u     = x0[6];
    v     = x0[7];
    w     = x0[8];
    p     = x0[9];
    q     = x0[10];
    r     = x0[11];

    x0dot[0] = cos(theta) * cos(psi) * u + (sin(phi) * sin(theta) * cos(psi) - cos(phi) * sin(psi)) * v +
               (cos(phi) * sin(theta) * cos(psi) + sin(phi) * sin(psi)) * w;

    x0dot[1] = cos(theta) * sin(psi) * u + (sin(phi) * sin(theta) * sin(psi) + cos(phi) * cos(psi)) * v +
               (cos(phi) * sin(theta) * sin(psi) - sin(phi) * cos(psi)) * w;

    x0dot[2] = -sin(theta) * u + sin(phi) * cos(theta) * v + cos(phi) * cos(theta) * w;

    x0dot[3] = p + (q * sin(phi) + r * cos(phi)) * tan(theta);

    x0dot[4] = q * cos(phi) - r * sin(phi);

    x0dot[5] = (q * sin(phi) + r * cos(phi)) / cos(theta);

    float XYZ = -NU * sqrt(u * u + v * v + w * w);
    float X   = XYZ * u;
    x0dot[6]  = (r * v - q * w) - GRAVITY * sin(theta) + MASS_INV * X;

    float Y  = XYZ * v;
    x0dot[7] = (p * w - r * u) + GRAVITY * cos(theta) * sin(phi) + MASS_INV * Y;

    float Z  = XYZ * w;
    x0dot[8] = (q * u - p * v) + GRAVITY * cos(theta) * cos(phi) + MASS_INV * Z + MASS_INV * Zc;

    float LMN = -MU * sqrt(p * p + q * q + r * r);
    float L   = LMN * p;
    x0dot[9]  = (IY - IZ) / IX * q * r + (1 / IX) * L + (1 / IX) * Lc;

    float M   = LMN * q;
    x0dot[10] = (IZ - IX) / IY * p * r + (1 / IY) * M + (1 / IY) * Mc;

    float N   = LMN * r;
    x0dot[11] = (IX - IY) / IZ * p * q + (1 / IZ) * N + (1 / IZ) * Nc;
}

/***************************/
/* GET PROPAGATION FUNCTION */
/***************************/
__device__ PropagateAndCheckFunc getPropagateAndCheckFunc()
{
    switch(MODEL)
        {
            case 0:
                return propagateAndCheckUnicycle;
            case 1:
                return propagateAndCheckDoubleIntRungeKutta;
            case 2:
                return propagateAndCheckDubinsAirplaneRungeKutta;
            case 3:
                return propagateAndCheckQuadRungeKutta;
            default:
                return nullptr;
        }
}