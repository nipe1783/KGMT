#include "hip/hip_runtime.h"

#include "statePropagator/statePropagator.cuh"

__device__ bool propagateAndCheck(float* x0, float* x1, hiprandState* seed, float* obstacles, int obstaclesCount)
{
    PropagateAndCheckFunc func = getPropagateAndCheckFunc();
    return func ? func(x0, x1, seed, obstacles, obstaclesCount) : false;
}

/***************************/
/* UNICYCLE PROPAGATION FUNCTION */
/***************************/
__device__ bool propagateAndCheckUnicycle(float* x0, float* x1, hiprandState* seed, float* obstacles, int obstaclesCount)
{
    float a        = UNI_MIN_ACC + hiprand_uniform(seed) * (UNI_MAX_ACC - UNI_MIN_ACC);
    float steering = UNI_MIN_STEERING + hiprand_uniform(seed) * (UNI_MAX_STEERING - UNI_MIN_STEERING);
    float duration = UNI_MIN_DT + hiprand_uniform(seed) * (UNI_MAX_DT - UNI_MIN_DT);
    float dt       = duration / NUM_DISC;

    float x     = x0[0];
    float y     = x0[1];
    float theta = x0[2];
    float v     = x0[3];

    float cosTheta, sinTheta, tanSteering;
    float bbMin[DIM], bbMax[DIM];

    bool motionValid = true;
    for(int i = 0; i < NUM_DISC; i++)
        {
            float x0State[DIM] = {x, y};
            cosTheta           = cos(theta);
            sinTheta           = sin(theta);
            tanSteering        = tan(steering);

            // --- State Propagation ---
            x += v * cosTheta * dt;
            y += v * sinTheta * dt;
            theta += (v / UNI_LENGTH) * tanSteering * dt;
            v += a * dt;
            float x1State[DIM] = {x, y};

            // --- Workspace Limit Check ---
            if(x < 0 || x > WS_SIZE || y < 0 || y > WS_SIZE)
                {
                    motionValid = false;
                    break;
                }

            // --- Obstacle Collision Check ---
            for(int d = 0; d < DIM; d++)
                {
                    if(x0State[d] > x1State[d])
                        {
                            bbMin[d] = x1State[d];
                            bbMax[d] = x0State[d];
                        }
                    else
                        {
                            bbMin[d] = x0State[d];
                            bbMax[d] = x1State[d];
                        }
                }

            motionValid = motionValid && isMotionValid(x0State, x1State, bbMin, bbMax, obstacles, obstaclesCount);
            if(!motionValid) break;
        }

    x1[0] = x, x1[1] = y, x1[2] = theta, x1[3] = v, x1[4] = a, x1[5] = steering, x1[6] = duration;
    return motionValid;
}

/***************************/
/* DOUBLE INTEGRATOR PROPAGATION FUNCTION */
/***************************/
__device__ bool propagateAndCheckDoubleInt(float* x0, float* x1, hiprandState* seed, float* obstacles, int obstaclesCount)
{
    float ax       = DI_MIN_ACC + hiprand_uniform(seed) * (DI_MAX_ACC - DI_MIN_ACC);
    float ay       = DI_MIN_ACC + hiprand_uniform(seed) * (DI_MAX_ACC - DI_MIN_ACC);
    float az       = DI_MIN_ACC + hiprand_uniform(seed) * (DI_MAX_ACC - DI_MIN_ACC);
    float duration = DI_MIN_DT + hiprand_uniform(seed) * (DI_MAX_DT - DI_MIN_DT);
    float dt       = duration / NUM_DISC;

    float x  = x0[0];
    float y  = x0[1];
    float z  = x0[2];
    float vx = x0[3];
    float vy = x0[4];
    float vz = x0[5];

    bool motionValid = true;
    float bbMin[DIM], bbMax[DIM];
    for(int i = 0; i < NUM_DISC; i++)
        {
            float x0State[DIM] = {x, y, z};

            // --- State Propagation ---
            x += vx * dt;
            y += vy * dt;
            z += vz * dt;
            vx += ax * dt;
            vy += ay * dt;
            vz += az * dt;
            float x1State[DIM] = {x, y, z};

            // --- Workspace Limit Check ---
            if(x < 0 || x > WS_SIZE || y < 0 || y > WS_SIZE || z < 0 || z > WS_SIZE)
                {
                    motionValid = false;
                    break;
                }

            // --- Obstacle Collision Check ---
            for(int d = 0; d < DIM; d++)
                {
                    if(x0State[d] > x1State[d])
                        {
                            bbMin[d] = x1State[d];
                            bbMax[d] = x0State[d];
                        }
                    else
                        {
                            bbMin[d] = x0State[d];
                            bbMax[d] = x1State[d];
                        }
                }

            motionValid = motionValid && isMotionValid(x0State, x1State, bbMin, bbMax, obstacles, obstaclesCount);
            if(!motionValid) break;
        }

    x1[0] = x, x1[1] = y, x1[2] = z, x1[3] = vx, x1[4] = vy, x1[5] = vz, x1[6] = ax, x1[7] = ay, x1[8] = az, x1[9] = duration;
    return motionValid;
}

/***************************/
/* GET PROPAGATION FUNCTION */
/***************************/
__device__ PropagateAndCheckFunc getPropagateAndCheckFunc()
{
    switch(MODEL)
        {
            case 0:
                return propagateAndCheckUnicycle;
            case 1:
                return propagateAndCheckDoubleInt;
            default:
                return nullptr;
        }
}