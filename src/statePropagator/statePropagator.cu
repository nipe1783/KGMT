#include "hip/hip_runtime.h"

#include "statePropagator/statePropagator.cuh"

__device__ bool propagateAndCheck(float* x0, float* x1, hiprandState* seed, float* obstacles, int obstaclesCount)
{
    PropagateAndCheckFunc func = getPropagateAndCheckFunc();
    return func ? func(x0, x1, seed, obstacles, obstaclesCount) : false;
}

/***************************/
/* UNICYCLE PROPAGATION FUNCTION */
/***************************/
__device__ bool propagateAndCheckUnicycle(float* x0, float* x1, hiprandState* seed, float* obstacles, int obstaclesCount)
{
    float a                 = A_MIN + hiprand_uniform(seed) * (A_MAX - A_MIN);
    float steering          = UNI_MIN_STEERING + hiprand_uniform(seed) * (UNI_MAX_STEERING - UNI_MIN_STEERING);
    float duration          = UNI_MIN_DT + hiprand_uniform(seed) * (UNI_MAX_DT - UNI_MIN_DT);
    int propagationDuration = 1 + (int)(hiprand_uniform(seed) * (MAX_PROPAGATION_DURATION));

    float x     = x0[0];
    float y     = x0[1];
    float theta = x0[2];
    float v     = x0[3];

    float cosTheta, sinTheta, tanSteering;
    float bbMin[W_DIM], bbMax[W_DIM];

    bool motionValid = true;
    for(int i = 0; i < propagationDuration; i++)
        {
            float x0State[W_DIM] = {x, y};
            cosTheta             = cos(theta);
            sinTheta             = sin(theta);
            tanSteering          = tan(steering);

            // --- State Propagation ---
            x += v * cosTheta * STEP_SIZE;
            y += v * sinTheta * STEP_SIZE;
            theta += (v / UNI_LENGTH) * tanSteering * STEP_SIZE;
            v += a * STEP_SIZE;
            float x1State[W_DIM] = {x, y};

            // --- Workspace Limit Check ---
            if(x < 0 || x > W_SIZE || y < 0 || y > W_SIZE)
                {
                    motionValid = false;
                    break;
                }

            // --- Obstacle Collision Check ---
            for(int d = 0; d < W_DIM; d++)
                {
                    if(x0State[d] > x1State[d])
                        {
                            bbMin[d] = x1State[d];
                            bbMax[d] = x0State[d];
                        }
                    else
                        {
                            bbMin[d] = x0State[d];
                            bbMax[d] = x1State[d];
                        }
                }

            motionValid = motionValid && isMotionValid(x0State, x1State, bbMin, bbMax, obstacles, obstaclesCount);
            if(!motionValid) break;
        }

    x1[0] = x, x1[1] = y, x1[2] = theta, x1[3] = v, x1[4] = a, x1[5] = steering, x1[6] = duration;
    return motionValid;
}

/***************************/
/* DOUBLE INTEGRATOR PROPAGATION FUNCTION */
/***************************/
__device__ bool propagateAndCheckDoubleIntRungeKutta(float* x0, float* x1, hiprandState* seed, float* obstacles, int obstaclesCount)
{
    float ax                = A_MIN + hiprand_uniform(seed) * (A_MAX - A_MIN);
    float ay                = A_MIN + hiprand_uniform(seed) * (A_MAX - A_MIN);
    float az                = A_MIN + hiprand_uniform(seed) * (A_MAX - A_MIN);
    int propagationDuration = 1 + (int)(hiprand_uniform(seed) * (MAX_PROPAGATION_DURATION));

    float x  = x0[0];
    float y  = x0[1];
    float z  = x0[2];
    float vx = x0[3];
    float vy = x0[4];
    float vz = x0[5];

    bool motionValid = true;
    float bbMin[W_DIM], bbMax[W_DIM];
    for(int i = 0; i < propagationDuration; i++)
        {
            float x0State[W_DIM] = {x, y, z};

            // --- State Propagation. 4th order Runge Kutta ---
            x += (vx + (vx + 2 * (vx + ax * STEP_SIZE / 2) + (vx + ax * STEP_SIZE))) * STEP_SIZE / 6;
            y += (vy + (vy + 2 * (vy + ay * STEP_SIZE / 2) + (vy + ay * STEP_SIZE))) * STEP_SIZE / 6;
            z += (vz + (vz + 2 * (vz + az * STEP_SIZE / 2) + (vz + az * STEP_SIZE))) * STEP_SIZE / 6;
            vx += (ax + 2 * ax + 2 * ax + ax) * STEP_SIZE / 6;
            vy += (ay + 2 * ay + 2 * ay + ay) * STEP_SIZE / 6;
            vz += (az + 2 * az + 2 * az + az) * STEP_SIZE / 6;

            // --- Dyanmics Validity Check ---
            if(vx < V_MIN || vx > V_MAX || vy < V_MIN || vy > V_MAX || vz < V_MIN || vz > V_MAX)
                {
                    motionValid = false;
                    break;
                }

            float x1State[W_DIM] = {x, y, z};

            // --- Workspace Limit Check ---
            if(x < 0 || x > W_SIZE || y < 0 || y > W_SIZE || z < 0 || z > W_SIZE)
                {
                    motionValid = false;
                    break;
                }

            // --- Obstacle Collision Check ---
            for(int d = 0; d < W_DIM; d++)
                {
                    if(x0State[d] > x1State[d])
                        {
                            bbMin[d] = x1State[d];
                            bbMax[d] = x0State[d];
                        }
                    else
                        {
                            bbMin[d] = x0State[d];
                            bbMax[d] = x1State[d];
                        }
                }

            motionValid = motionValid && isMotionValid(x0State, x1State, bbMin, bbMax, obstacles, obstaclesCount);
            if(!motionValid) break;
        }

    x1[0] = x, x1[1] = y, x1[2] = z, x1[3] = vx, x1[4] = vy, x1[5] = vz, x1[6] = ax, x1[7] = ay, x1[8] = az,
    x1[9] = STEP_SIZE * propagationDuration;
    return motionValid;
}

/***************************/
/* DUBINS AIRPLANE PROPAGATION FUNCTION */
/***************************/
__device__ bool propagateAndCheckDubinsAirplaneRungeKutta(float* x0, float* x1, hiprandState* seed, float* obstacles, int obstaclesCount)
{
    float a                 = A_MIN + hiprand_uniform(seed) * (A_MAX - A_MIN);
    float yawRate           = DUBINS_AIRPLANE_MIN_YR + hiprand_uniform(seed) * (DUBINS_AIRPLANE_MAX_YR - DUBINS_AIRPLANE_MIN_YR);
    float pitchRate         = DUBINS_AIRPLANE_MIN_PR + hiprand_uniform(seed) * (DUBINS_AIRPLANE_MAX_PR - DUBINS_AIRPLANE_MIN_PR);
    int propagationDuration = 1 + (int)(hiprand_uniform(seed) * (MAX_PROPAGATION_DURATION));

    float x     = x0[0];
    float y     = x0[1];
    float z     = x0[2];
    float yaw   = x0[3];
    float pitch = x0[4];
    float v     = x0[5];

    bool motionValid = true;
    float bbMin[W_DIM], bbMax[W_DIM];

    for(int i = 0; i < propagationDuration; i++)
        {
            float x0State[W_DIM] = {x, y, z};

            // --- State Propagation using 4th Order Runge-Kutta Method ---
            x +=
              (STEP_SIZE / 6.0f) *
              (v * cosf(pitch) * cosf(yaw) +
               2.0f * ((v + 0.5f * STEP_SIZE * a) * cosf(pitch + 0.5f * STEP_SIZE * pitchRate) * cosf(yaw + 0.5f * STEP_SIZE * yawRate) +
                       (v + 0.5f * STEP_SIZE * a) * cosf(pitch + 0.5f * STEP_SIZE * pitchRate) * cosf(yaw + 0.5f * STEP_SIZE * yawRate)) +
               (v + STEP_SIZE * a) * cosf(pitch + STEP_SIZE * pitchRate) * cosf(yaw + STEP_SIZE * yawRate));
            y +=
              (STEP_SIZE / 6.0f) *
              (v * cosf(pitch) * sinf(yaw) +
               2.0f * ((v + 0.5f * STEP_SIZE * a) * cosf(pitch + 0.5f * STEP_SIZE * pitchRate) * sinf(yaw + 0.5f * STEP_SIZE * yawRate) +
                       (v + 0.5f * STEP_SIZE * a) * cosf(pitch + 0.5f * STEP_SIZE * pitchRate) * sinf(yaw + 0.5f * STEP_SIZE * yawRate)) +
               (v + STEP_SIZE * a) * cosf(pitch + STEP_SIZE * pitchRate) * sinf(yaw + STEP_SIZE * yawRate));
            z += (STEP_SIZE / 6.0f) * (v * sinf(pitch) +
                                       2.0f * ((v + 0.5f * STEP_SIZE * a) * sinf(pitch + 0.5f * STEP_SIZE * pitchRate) +
                                               (v + 0.5f * STEP_SIZE * a) * sinf(pitch + 0.5f * STEP_SIZE * pitchRate)) +
                                       (v + STEP_SIZE * a) * sinf(pitch + STEP_SIZE * pitchRate));
            yaw += STEP_SIZE * yawRate;
            pitch += STEP_SIZE * pitchRate;
            v += (STEP_SIZE / 6.0f) * (a + 2.0f * (a + a) + a);

            // --- Dynamics Validity Check ---'
            if(v < V_MIN || v > V_MAX)
                {
                    motionValid = false;
                    break;
                }
            else if(pitch < DUBINS_AIRPLANE_MIN_PITCH || pitch > DUBINS_AIRPLANE_MAX_PITCH)
                {
                    motionValid = false;
                    break;
                }

            float x1State[W_DIM] = {x, y, z};

            // --- Workspace Limit Check ---
            if(x < 0 || x > W_SIZE || y < 0 || y > W_SIZE || z < 0 || z > W_SIZE)
                {
                    motionValid = false;
                    break;
                }

            // --- Obstacle Collision Check ---
            for(int d = 0; d < W_DIM; d++)
                {
                    if(x0State[d] > x1State[d])
                        {
                            bbMin[d] = x1State[d];
                            bbMax[d] = x0State[d];
                        }
                    else
                        {
                            bbMin[d] = x0State[d];
                            bbMax[d] = x1State[d];
                        }
                }

            motionValid = motionValid && isMotionValid(x0State, x1State, bbMin, bbMax, obstacles, obstaclesCount);
            if(!motionValid) break;
        }

    x1[0] = x;
    x1[1] = y;
    x1[2] = z;
    x1[3] = yaw;
    x1[4] = pitch;
    x1[5] = v;
    x1[6] = yawRate;
    x1[7] = pitchRate;
    x1[8] = a;
    x1[9] = STEP_SIZE * propagationDuration;

    return motionValid;
}

/***************************/
/* QUAD PROPAGATION FUNCTION */
/***************************/
__device__ bool propagateAndCheckQuadRungeKutta(float* x0, float* x1, hiprandState* seed, float* obstacles, int obstaclesCount)
{
    float Zc = QUAD_MIN_Zc + hiprand_uniform(seed) * (QUAD_MAX_Zc - QUAD_MIN_Zc);
    float Lc = QUAD_MIN_Lc + hiprand_uniform(seed) * (QUAD_MAX_Lc - QUAD_MIN_Lc);
    float Mc = QUAD_MIN_Mc + hiprand_uniform(seed) * (QUAD_MAX_Mc - QUAD_MIN_Mc);
    float Nc = QUAD_MIN_Nc + hiprand_uniform(seed) * (QUAD_MAX_Nc - QUAD_MIN_Nc);

    int propagationDuration = 1 + (int)(hiprand_uniform(seed) * (MAX_PROPAGATION_DURATION));

    float x     = x0[0];
    float y     = x0[1];
    float z     = x0[2];
    float phi   = x0[3];
    float theta = x0[4];
    float psi   = x0[5];
    float u     = x0[6];
    float v     = x0[7];
    float w     = x0[8];
    float p     = x0[9];
    float q     = x0[10];
    float r     = x0[11];

    bool motionValid = true;
    float bbMin[W_DIM], bbMax[W_DIM];

    float h1[12];
    float h2[12];
    float h3[12];
    float h4[12];

    for(int i = 0; i < propagationDuration; i++)
        {
            float x0State[W_DIM] = {x, y, z};

            ode(h1, x0, Zc, Lc, Mc, Nc);
            ode(h2, h1, Zc, Lc, Mc, Nc);
            ode(h3, h2, Zc, Lc, Mc, Nc);
            ode(h4, h3, Zc, Lc, Mc, Nc);

            x += STEP_SIZE / 6 * (h1[0] + 2.0f * h2[0] + 2.0f * h3[0] + h4[0]);
            y += STEP_SIZE / 6 * (h1[1] + 2.0f * h2[1] + 2.0f * h3[1] + h4[1]);
            z += STEP_SIZE / 6 * (h1[2] + 2.0f * h2[2] + 2.0f * h3[2] + h4[2]);
            phi += STEP_SIZE / 6 * (h1[3] + 2.0f * h2[3] + 2.0f * h3[3] + h4[3]);
            theta += STEP_SIZE / 6 * (h1[4] + 2.0f * h2[4] + 2.0f * h3[4] + h4[4]);
            psi += STEP_SIZE / 6 * (h1[5] + 2.0f * h2[5] + 2.0f * h3[5] + h4[5]);
            u += STEP_SIZE / 6 * (h1[6] + 2.0f * h2[6] + 2.0f * h3[6] + h4[6]);
            v += STEP_SIZE / 6 * (h1[7] + 2.0f * h2[7] + 2.0f * h3[7] + h4[7]);
            w += STEP_SIZE / 6 * (h1[8] + 2.0f * h2[8] + 2.0f * h3[8] + h4[8]);
            p += STEP_SIZE / 6 * (h1[9] + 2.0f * h2[9] + 2.0f * h3[9] + h4[9]);
            q += STEP_SIZE / 6 * (h1[10] + 2.0f * h2[10] + 2.0f * h3[10] + h4[10]);
            r += STEP_SIZE / 6 * (h1[11] + 2.0f * h2[11] + 2.0f * h3[11] + h4[11]);

            float x1State[W_DIM] = {x, y, z};

            // --- Vehicle Dynamics Check ---
            if(u < V_MIN || u > V_MAX || v < V_MIN || v > V_MAX || w < V_MIN || w > V_MAX)
                {
                    motionValid = false;
                    break;
                }

            // --- Workspace Limit Check ---
            if(x < W_MIN || x > W_MAX || y < W_MIN || y > W_MAX || z < W_MIN || z > W_MAX)
                {
                    motionValid = false;
                    break;
                }

            // --- Obstacle Collision Check ---
            for(int d = 0; d < W_DIM; d++)
                {
                    if(x0State[d] > x1State[d])
                        {
                            bbMin[d] = x1State[d];
                            bbMax[d] = x0State[d];
                        }
                    else
                        {
                            bbMin[d] = x0State[d];
                            bbMax[d] = x1State[d];
                        }
                }

            motionValid = motionValid && isMotionValid(x0State, x1State, bbMin, bbMax, obstacles, obstaclesCount);
            if(!motionValid) break;
        }

    x1[0]  = x;
    x1[1]  = y;
    x1[2]  = z;
    x1[3]  = phi;
    x1[4]  = theta;
    x1[5]  = psi;
    x1[6]  = u;
    x1[7]  = v;
    x1[8]  = w;
    x1[9]  = p;
    x1[10] = q;
    x1[11] = r;
    x1[12] = Zc;
    x1[13] = Lc;
    x1[14] = Mc;
    x1[15] = Nc;
    x1[16] = STEP_SIZE * propagationDuration;

    return motionValid;
}

__device__ void ode(float* x0dot, float* x0, float Zc, float Lc, float Mc, float Nc)
{
    float phi, theta, psi, u, v, w, p, q, r;
    phi   = x0[3];
    theta = x0[4];
    psi   = x0[5];
    u     = x0[6];
    v     = x0[7];
    w     = x0[8];
    p     = x0[9];
    q     = x0[10];
    r     = x0[11];

    x0dot[0] = cos(theta) * cos(psi) * u + (sin(phi) * sin(theta) * cos(psi) - cos(phi) * sin(psi)) * v +
               (cos(phi) * sin(theta) * cos(psi) + sin(phi) * sin(psi)) * w;

    x0dot[1] = cos(theta) * sin(psi) * u + (sin(phi) * sin(theta) * sin(psi) + cos(phi) * cos(psi)) * v +
               (cos(phi) * sin(theta) * sin(psi) - sin(phi) * cos(psi)) * w;

    x0dot[2] = -sin(theta) * u + sin(phi) * cos(theta) * v + cos(phi) * cos(theta) * w;

    x0dot[3] = p + (q * sin(phi) + r * cos(phi)) * tan(theta);

    x0dot[4] = q * cos(phi) - r * sin(phi);

    x0dot[5] = (q * sin(phi) + r * cos(phi)) / cos(theta);

    float XYZ = -NU * sqrt(u * u + v * v + w * w);
    float X   = XYZ * u;
    x0dot[6]  = (r * v - q * w) - GRAVITY * sin(theta) + MASS_INV * X;

    float Y  = XYZ * v;
    x0dot[7] = (p * w - r * u) + GRAVITY * cos(theta) * sin(phi) + MASS_INV * Y;

    float Z  = XYZ * w;
    x0dot[8] = (q * u - p * v) + GRAVITY * cos(theta) * cos(phi) + MASS_INV * Z + MASS_INV * Zc;

    float LMN = -MU * sqrt(p * p + q * q + r * r);
    float L   = LMN * p;
    x0dot[9]  = (IY - IZ) / IX * q * r + (1 / IX) * L + (1 / IX) * Lc;

    float M   = LMN * q;
    x0dot[10] = (IZ - IX) / IY * p * r + (1 / IY) * M + (1 / IY) * Mc;

    float N   = LMN * r;
    x0dot[11] = (IX - IY) / IZ * p * q + (1 / IZ) * N + (1 / IZ) * Nc;
}

/***************************/
/* GET PROPAGATION FUNCTION */
/***************************/
__device__ PropagateAndCheckFunc getPropagateAndCheckFunc()
{
    switch(MODEL)
        {
            case 0:
                return propagateAndCheckUnicycle;
            case 1:
                return propagateAndCheckDoubleIntRungeKutta;
            case 2:
                return propagateAndCheckDubinsAirplaneRungeKutta;
            case 3:
                return propagateAndCheckQuadRungeKutta;
            default:
                return nullptr;
        }
}