#include "hip/hip_runtime.h"

#include "statePropagator/statePropagator.cuh"

__device__ bool propagateAndCheck(float* x0, float* x1, hiprandState* seed, float* obstacles, int obstaclesCount)
{
    PropagateAndCheckFunc func = getPropagateAndCheckFunc();
    return func ? func(x0, x1, seed, obstacles, obstaclesCount) : false;
}

/***************************/
/* UNICYCLE PROPAGATION FUNCTION */
/***************************/
__device__ bool propagateAndCheckUnicycle(float* x0, float* x1, hiprandState* seed, float* obstacles, int obstaclesCount)
{
    float a                 = UNI_MIN_ACC + hiprand_uniform(seed) * (UNI_MAX_ACC - UNI_MIN_ACC);
    float steering          = UNI_MIN_STEERING + hiprand_uniform(seed) * (UNI_MAX_STEERING - UNI_MIN_STEERING);
    float duration          = UNI_MIN_DT + hiprand_uniform(seed) * (UNI_MAX_DT - UNI_MIN_DT);
    int propagationDuration = 1 + (int)(hiprand_uniform(seed) * (MAX_PROPAGATION_DURATION));

    float x     = x0[0];
    float y     = x0[1];
    float theta = x0[2];
    float v     = x0[3];

    float cosTheta, sinTheta, tanSteering;
    float bbMin[DIM], bbMax[DIM];

    bool motionValid = true;
    for(int i = 0; i < propagationDuration; i++)
        {
            float x0State[DIM] = {x, y};
            cosTheta           = cos(theta);
            sinTheta           = sin(theta);
            tanSteering        = tan(steering);

            // --- State Propagation ---
            x += v * cosTheta * STEP_SIZE;
            y += v * sinTheta * STEP_SIZE;
            theta += (v / UNI_LENGTH) * tanSteering * STEP_SIZE;
            v += a * STEP_SIZE;
            float x1State[DIM] = {x, y};

            // --- Workspace Limit Check ---
            if(x < 0 || x > WS_SIZE || y < 0 || y > WS_SIZE)
                {
                    motionValid = false;
                    break;
                }

            // --- Obstacle Collision Check ---
            for(int d = 0; d < DIM; d++)
                {
                    if(x0State[d] > x1State[d])
                        {
                            bbMin[d] = x1State[d];
                            bbMax[d] = x0State[d];
                        }
                    else
                        {
                            bbMin[d] = x0State[d];
                            bbMax[d] = x1State[d];
                        }
                }

            motionValid = motionValid && isMotionValid(x0State, x1State, bbMin, bbMax, obstacles, obstaclesCount);
            if(!motionValid) break;
        }

    x1[0] = x, x1[1] = y, x1[2] = theta, x1[3] = v, x1[4] = a, x1[5] = steering, x1[6] = duration;
    return motionValid;
}

/***************************/
/* DOUBLE INTEGRATOR PROPAGATION FUNCTION */
/***************************/
__device__ bool propagateAndCheckDoubleIntEular(float* x0, float* x1, hiprandState* seed, float* obstacles, int obstaclesCount)
{
    float ax                = DI_MIN_ACC + hiprand_uniform(seed) * (DI_MAX_ACC - DI_MIN_ACC);
    float ay                = DI_MIN_ACC + hiprand_uniform(seed) * (DI_MAX_ACC - DI_MIN_ACC);
    float az                = DI_MIN_ACC + hiprand_uniform(seed) * (DI_MAX_ACC - DI_MIN_ACC);
    float duration          = DI_MIN_DT + hiprand_uniform(seed) * (DI_MAX_DT - DI_MIN_DT);
    int propagationDuration = 1 + (int)(hiprand_uniform(seed) * (MAX_PROPAGATION_DURATION));

    float x  = x0[0];
    float y  = x0[1];
    float z  = x0[2];
    float vx = x0[3];
    float vy = x0[4];
    float vz = x0[5];

    bool motionValid = true;
    float bbMin[DIM], bbMax[DIM];
    for(int i = 0; i < propagationDuration; i++)
        {
            float x0State[DIM] = {x, y, z};

            // --- State Propagation. Eular Method ---
            x += vx * STEP_SIZE;
            y += vy * STEP_SIZE;
            z += vz * STEP_SIZE;
            vx += ax * STEP_SIZE;
            vy += ay * STEP_SIZE;
            vz += az * STEP_SIZE;
            float x1State[DIM] = {x, y, z};

            // --- Workspace Limit Check ---
            if(x < 0 || x > WS_SIZE || y < 0 || y > WS_SIZE || z < 0 || z > WS_SIZE)
                {
                    motionValid = false;
                    break;
                }

            // --- Obstacle Collision Check ---
            for(int d = 0; d < DIM; d++)
                {
                    if(x0State[d] > x1State[d])
                        {
                            bbMin[d] = x1State[d];
                            bbMax[d] = x0State[d];
                        }
                    else
                        {
                            bbMin[d] = x0State[d];
                            bbMax[d] = x1State[d];
                        }
                }

            motionValid = motionValid && isMotionValid(x0State, x1State, bbMin, bbMax, obstacles, obstaclesCount);
            if(!motionValid) break;
        }

    x1[0] = x, x1[1] = y, x1[2] = z, x1[3] = vx, x1[4] = vy, x1[5] = vz, x1[6] = ax, x1[7] = ay, x1[8] = az, x1[9] = duration;
    return motionValid;
}

/***************************/
/* DOUBLE INTEGRATOR PROPAGATION FUNCTION */
/***************************/
__device__ bool propagateAndCheckDoubleIntRungeKutta(float* x0, float* x1, hiprandState* seed, float* obstacles, int obstaclesCount)
{
    float ax                = DI_MIN_ACC + hiprand_uniform(seed) * (DI_MAX_ACC - DI_MIN_ACC);
    float ay                = DI_MIN_ACC + hiprand_uniform(seed) * (DI_MAX_ACC - DI_MIN_ACC);
    float az                = DI_MIN_ACC + hiprand_uniform(seed) * (DI_MAX_ACC - DI_MIN_ACC);
    int propagationDuration = 1 + (int)(hiprand_uniform(seed) * (MAX_PROPAGATION_DURATION));

    float x  = x0[0];
    float y  = x0[1];
    float z  = x0[2];
    float vx = x0[3];
    float vy = x0[4];
    float vz = x0[5];

    bool motionValid = true;
    float bbMin[DIM], bbMax[DIM];
    for(int i = 0; i < propagationDuration; i++)
        {
            float x0State[DIM] = {x, y, z};

            // --- State Propagation. 4th order Runge Kutta ---
            x += (vx + (vx + 2 * (vx + ax * STEP_SIZE / 2) + (vx + ax * STEP_SIZE))) * STEP_SIZE / 6;
            y += (vy + (vy + 2 * (vy + ay * STEP_SIZE / 2) + (vy + ay * STEP_SIZE))) * STEP_SIZE / 6;
            z += (vz + (vz + 2 * (vz + az * STEP_SIZE / 2) + (vz + az * STEP_SIZE))) * STEP_SIZE / 6;
            vx += (ax + 2 * ax + 2 * ax + ax) * STEP_SIZE / 6;
            vy += (ay + 2 * ay + 2 * ay + ay) * STEP_SIZE / 6;
            vz += (az + 2 * az + 2 * az + az) * STEP_SIZE / 6;

            float x1State[DIM] = {x, y, z};

            // --- Workspace Limit Check ---
            if(x < 0 || x > WS_SIZE || y < 0 || y > WS_SIZE || z < 0 || z > WS_SIZE)
                {
                    motionValid = false;
                    break;
                }

            // --- Obstacle Collision Check ---
            for(int d = 0; d < DIM; d++)
                {
                    if(x0State[d] > x1State[d])
                        {
                            bbMin[d] = x1State[d];
                            bbMax[d] = x0State[d];
                        }
                    else
                        {
                            bbMin[d] = x0State[d];
                            bbMax[d] = x1State[d];
                        }
                }

            motionValid = motionValid && isMotionValid(x0State, x1State, bbMin, bbMax, obstacles, obstaclesCount);
            if(!motionValid) break;
        }

    x1[0] = x, x1[1] = y, x1[2] = z, x1[3] = vx, x1[4] = vy, x1[5] = vz, x1[6] = ax, x1[7] = ay, x1[8] = az,
    x1[9] = STEP_SIZE * propagationDuration;
    return motionValid;
}

/***************************/
/* GET PROPAGATION FUNCTION */
/***************************/
__device__ PropagateAndCheckFunc getPropagateAndCheckFunc()
{
    switch(MODEL)
        {
            case 0:
                return propagateAndCheckUnicycle;
            case 1:
                return propagateAndCheckDoubleIntRungeKutta;
            default:
                return nullptr;
        }
}