#include "hip/hip_runtime.h"
#include "graphs/Graph.cuh"
#include "config/config.h"

Graph::Graph(const float ws)
{
    h_r1Size_ = ws / R1;
    h_r2Size_ = ws / (R1 * R2);
    h_numEdges_ = (DIM == 2) ? pow(R1, 2) * 4 : pow(R1, 3) * 6;
    h_vertexArray_.resize((DIM == 2) ? pow(R1, 2) : pow(R1, 3));
    constructVertexArray();
    constructEdgeArray();
    constructFromVertices();
    constructToVertices();
    if(VERBOSE)
        {
            printf("/***************************/\n");
            printf("/* Graph Dimension: %d */\n", DIM);
            printf("/* Number of Edges: %d */\n", h_numEdges_);
            printf("/***************************/\n");
        }
    // TODO: Delete Comments
    // if(VERBOSE)
    //     {
    //         printf("/***************************/\n");
    //         printf("/* Vertex Array */\n");
    //         for(int i = 0; i < h_vertexArray_.size(); ++i)
    //             {
    //                 printf("Vertex %d: %d\n", i, h_vertexArray_[i]);
    //             }
    //         printf("/***************************/\n");
    //     }

    // if(VERBOSE)
    //     {
    //         printf("/***************************/\n");
    //         printf("/* Edge Array */\n");
    //         for(int i = 0; i < h_edgeArray_.size(); ++i)
    //             {
    //                 printf("Edge %d: %d\n", i, h_edgeArray_[i]);
    //             }
    //         printf("/***************************/\n");
    //     }

    // if(VERBOSE)
    //     {
    //         printf("/***************************/\n");
    //         printf("/* From->To Vertices */\n");
    //         for(int i = 0; i < h_fromVertices_.size(); ++i)
    //             {
    //                 printf("From Vertex %d: To Vertex %d\n", h_fromVertices_[i], h_toVertices_[i]);
    //             }
    //     }
}

void Graph::constructVertexArray()
{
    int edgeIdx = 0;
    for(int i = 0; i < R1; ++i)
        {
            for(int j = 0; j < R1; ++j)
                {
                    for(int k = 0; k < (DIM == 3 ? R1 : 1); ++k)
                        {
                            int currentNode = (DIM == 2) ? i * R1 + j : (i * R1 * R1) + (j * R1) + k;
                            h_vertexArray_[currentNode] = edgeIdx;

                            // Calculate the number of edges for the current node
                            int edges = 0;
                            if(DIM == 2)
                                {
                                    if(i > 0) edges++;
                                    if(j > 0) edges++;
                                    if(i < R1 - 1) edges++;
                                    if(j < R1 - 1) edges++;
                                }
                            else if(DIM == 3)
                                {
                                    if(i > 0) edges++;
                                    if(j > 0) edges++;
                                    if(k > 0) edges++;
                                    if(i < R1 - 1) edges++;
                                    if(j < R1 - 1) edges++;
                                    if(k < R1 - 1) edges++;
                                }

                            edgeIdx += edges;
                        }
                }
        }
}

void Graph::constructEdgeArray()
{
    for(int i = 0; i < R1; ++i)
        {
            for(int j = 0; j < R1; ++j)
                {
                    for(int k = 0; k < (DIM == 3 ? R1 : 1); ++k)
                        {
                            if(DIM == 2)
                                {
                                    if(i > 0) h_edgeArray_.push_back((i - 1) * R1 + j);
                                    if(j > 0) h_edgeArray_.push_back(i * R1 + (j - 1));
                                    if(i < R1 - 1) h_edgeArray_.push_back((i + 1) * R1 + j);
                                    if(j < R1 - 1) h_edgeArray_.push_back(i * R1 + (j + 1));
                                }
                            if(DIM == 3)
                                {
                                    if(i > 0) h_edgeArray_.push_back(((i - 1) * R1 * R1) + (j * R1) + k);
                                    if(j > 0) h_edgeArray_.push_back((i * R1 * R1) + ((j - 1) * R1) + k);
                                    if(k > 0) h_edgeArray_.push_back((i * R1 * R1) + (j * R1) + (k - 1));
                                    if(i < R1 - 1) h_edgeArray_.push_back(((i + 1) * R1 * R1) + (j * R1) + k);
                                    if(j < R1 - 1) h_edgeArray_.push_back((i * R1 * R1) + ((j + 1) * R1) + k);
                                    if(k < R1 - 1) h_edgeArray_.push_back((i * R1 * R1) + (j * R1) + (k + 1));
                                }
                        }
                }
        }
}

void Graph::constructFromVertices()
{
    for(int i = 0; i < R1; ++i)
        {
            for(int j = 0; j < R1; ++j)
                {
                    for(int k = 0; k < (DIM == 3 ? R1 : 1); ++k)
                        {
                            int currentVertex = (DIM == 2) ? i * R1 + j : (i * R1 * R1) + (j * R1) + k;
                            if(DIM == 2)
                                {
                                    if(i > 0) h_fromVertices_.push_back(currentVertex);
                                    if(j > 0) h_fromVertices_.push_back(currentVertex);
                                    if(i < R1 - 1) h_fromVertices_.push_back(currentVertex);
                                    if(j < R1 - 1) h_fromVertices_.push_back(currentVertex);
                                }
                            if(DIM == 3)
                                {
                                    if(i > 0) h_fromVertices_.push_back(currentVertex);
                                    if(j > 0) h_fromVertices_.push_back(currentVertex);
                                    if(k > 0) h_fromVertices_.push_back(currentVertex);
                                    if(i < R1 - 1) h_fromVertices_.push_back(currentVertex);
                                    if(j < R1 - 1) h_fromVertices_.push_back(currentVertex);
                                    if(k < R1 - 1) h_fromVertices_.push_back(currentVertex);
                                }
                        }
                }
        }
}

void Graph::constructToVertices()
{
    for(int i = 0; i < R1; ++i)
        {
            for(int j = 0; j < R1; ++j)
                {
                    for(int k = 0; k < (DIM == 3 ? R1 : 1); ++k)
                        {
                            if(DIM == 2)
                                {
                                    if(i > 0) h_toVertices_.push_back((i - 1) * R1 + j);
                                    if(j > 0) h_toVertices_.push_back(i * R1 + (j - 1));
                                    if(i < R1 - 1) h_toVertices_.push_back((i + 1) * R1 + j);
                                    if(j < R1 - 1) h_toVertices_.push_back(i * R1 + (j + 1));
                                }
                            if(DIM == 3)
                                {
                                    if(i > 0) h_toVertices_.push_back(((i - 1) * R1 * R1) + (j * R1) + k);
                                    if(j > 0) h_toVertices_.push_back((i * R1 * R1) + ((j - 1) * R1) + k);
                                    if(k > 0) h_toVertices_.push_back((i * R1 * R1) + (j * R1) + (k - 1));
                                    if(i < R1 - 1) h_toVertices_.push_back(((i + 1) * R1 * R1) + (j * R1) + k);
                                    if(j < R1 - 1) h_toVertices_.push_back((i * R1 * R1) + ((j + 1) * R1) + k);
                                    if(k < R1 - 1) h_toVertices_.push_back((i * R1 * R1) + (j * R1) + (k + 1));
                                }
                        }
                }
        }
}

__host__ __device__ int getVertex(float x, float y, float r1Size)
{
    int cellX = static_cast<int>(x / r1Size);
    int cellY = static_cast<int>(y / r1Size);

    if(cellX >= 0 && cellX < R1 && cellY >= 0 && cellY < R1)
        {
            return cellY * R1 + cellX;
        }
    return -1;
}

__host__ __device__ int getVertex(float x, float y, float z, float r1Size)
{
    int cellX = static_cast<int>(x / r1Size);
    int cellY = static_cast<int>(y / r1Size);
    int cellZ = static_cast<int>(z / r1Size);

    if(cellX >= 0 && cellX < R1 && cellY >= 0 && cellY < R1 && (cellZ >= 0 && cellZ < R1))
        {
            return (cellY * R1 + cellX) * R1 + cellZ;
        }
    return -1;
}

__host__ __device__ int getSubVertex(float x, float y, int r1, float r1Size, float r2Size)
{
    if(r1 == -1)
        {
            return -1;
        }

    int cellX_R2 = static_cast<int>((x - (r1 % R1) * r1Size) / r2Size);
    int cellY_R2 = static_cast<int>((y - (r1 / R1) * r1Size) / r2Size);
    if(cellX_R2 >= 0 && cellX_R2 < R2 && cellY_R2 >= 0 && cellY_R2 < R2)
        {
            return r1 * (R2 * R2) + (cellY_R2 * R2 + cellX_R2);
        }
    return -1;
}

__host__ __device__ int getSubVertex(float x, float y, float z, int r1, float r1Size, float r2Size)
{
    if(r1 == -1)
        {
            return -1;
        }

    int cellX_R2 = static_cast<int>((x - (r1 / (R1 * R1)) * r1Size) / r2Size);
    int cellY_R2 = static_cast<int>((y - ((r1 / R1) % R1) * r1Size) / r2Size);
    int cellZ_R2 = static_cast<int>((z - (r1 % R1) * r1Size) / r2Size);
    if(cellX_R2 >= 0 && cellX_R2 < R2 && cellY_R2 >= 0 && cellY_R2 < R2 && cellZ_R2 >= 0 && cellZ_R2 < R2)
        {
            return r1 * (R2 * R2 * R2) + ((cellY_R2 * R2 + cellX_R2) * R2 + cellZ_R2);
        }
    return -1;
}

__host__ __device__ int hashEdge(int key, int size)
{
    return key % size;
}

__host__ __device__ int getEdge(int fromVertex, int toVertex, int* hashTable, int numEdges)
{
    int key = fromVertex * 100000 + toVertex;
    int hash = hashEdge(key, numEdges);
    while(hashTable[2 * hash] != key)
        {
            if(hashTable[2 * hash] == -1)
                {
                    return -1;
                }
            hash = (hash + 1) % numEdges;
        }
    return hashTable[2 * hash + 1];
}