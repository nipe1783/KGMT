#include "hip/hip_runtime.h"
#include "planners/Planner.cuh"
#include "config/config.h"

Planner::Planner(int h_desiredTreeSize)
{
    h_desiredTreeSize_ = h_desiredTreeSize;
    h_maxTreeSize_     = h_desiredTreeSize;
    h_gridSize_        = iDivUp(h_maxTreeSize_, h_blockSize_);

    d_treeSamples_           = thrust::device_vector<float>(h_maxTreeSize_ * SAMPLE_DIM);
    d_treeSamplesParentIdxs_ = thrust::device_vector<int>(h_maxTreeSize_);
    d_treeSampleCosts_       = thrust::device_vector<float>(h_maxTreeSize_);
    d_controlPathToGoal_     = thrust::device_vector<float>(MAX_ITER * SAMPLE_DIM);
    d_randomSeeds_           = thrust::device_vector<hiprandState>(h_maxTreeSize_);

    d_treeSamples_ptr_           = thrust::raw_pointer_cast(d_treeSamples_.data());
    d_treeSamplesParentIdxs_ptr_ = thrust::raw_pointer_cast(d_treeSamplesParentIdxs_.data());
    d_treeSampleCosts_ptr_       = thrust::raw_pointer_cast(d_treeSampleCosts_.data());
    d_controlPathToGoal_ptr_     = thrust::raw_pointer_cast(d_controlPathToGoal_.data());
    d_randomSeeds_ptr_           = thrust::raw_pointer_cast(d_randomSeeds_.data());

    hipMalloc(&d_costToGoal_ptr_, sizeof(float));
    hipMalloc(&d_pathToGoal_ptr_, sizeof(int));

    h_controlPathToGoal_ = new float[SAMPLE_DIM * MAX_ITER];

    if(VERBOSE)
        {
            printf("/***************************/\n");
            printf("/* Workspace Dimension: %d */\n", W_DIM);
            printf("/* Workspace Size: %f */\n", W_SIZE);
            printf("/* Maximum discretization steps in propagation: %d */\n", MAX_PROPAGATION_DURATION);
            printf("/* Propagation step Size: %f */\n", STEP_SIZE);
            printf("/* Max Tree Size: %d */\n", h_maxTreeSize_);
            printf("/* Goal Distance Threshold: %f */\n", GOAL_THRESH);
            printf("/* Max Planning Iterations: %d */\n", MAX_ITER);
        }
}

__global__ void initializeRandomSeeds_kernel(hiprandState* randomSeeds, int numSeeds, int seed)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid < numSeeds)
        {
            hiprand_init(seed, tid, 0, &randomSeeds[tid]);
        }
}

void Planner::initializeRandomSeeds(int seed)
{
    int blockSize = 32;
    initializeRandomSeeds_kernel<<<iDivUp(h_maxTreeSize_, blockSize), blockSize>>>(d_randomSeeds_ptr_, h_maxTreeSize_, seed);
}

__global__ void findInd(uint numSamples, bool* S, uint* scanIdx, uint* activeS)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid >= numSamples) return;
    if(!S[tid]) return;
    activeS[scanIdx[tid]] = tid;
}

__global__ void findInd(uint numSamples, uint* S, uint* scanIdx, uint* activeS)
{
    int node = blockIdx.x * blockDim.x + threadIdx.x;
    if(node >= numSamples) return;
    if(!S[node]) return;
    activeS[scanIdx[node]] = node;
}

__global__ void repeatInd(uint numSamples, uint* activeS, uint* C, uint* prefixSum, uint* repeatedInd)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid >= numSamples) return;

    uint index    = activeS[tid];
    uint count    = C[index];
    uint startPos = prefixSum[index];
    for(uint i = 0; i < count; ++i)
        {
            if(startPos + i >= numSamples) return;
            repeatedInd[startPos + i] = index;
        }
}